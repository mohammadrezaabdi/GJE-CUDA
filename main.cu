#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include "matrix_utils.cpp"
#include <hip/hip_runtime.h>
#include <iomanip>

#define RANDOM 1
#define FROM_FILE 2

#define BLOCK_DIM 1<<10
#define COL_PER_BLK 3

#define min(a, b) ((a<b)?a:b)

#define MARGIN 1e-8
#define IS_ZERO(x) (abs(x)<=MARGIN)
#define NOT_ZERO(x) (abs(x)>MARGIN)


using namespace std;

__device__ void normalize_row(double *target_row, const double *base_row, double scale, size_t n, size_t offset) {
    for (size_t i = 0; i < n; i++) {
        double temp = target_row[i + offset] - (base_row[i] * scale);
        target_row[i + offset] = (IS_ZERO(temp) ? 0 : temp);
    }
}

__device__ void
normalize_self(double *self, double const *self_but_in_share_memory, double scale, size_t n, size_t offset) {
    for (size_t i = 0; i < n; i++) {
        self[i + offset] = self_but_in_share_memory[i] / scale;
    }
}

__global__ void gje_inverse(double *m2, size_t n, size_t base_row_index, double *scale) {
    size_t m2_width = 2 * n;
    extern __shared__ double base_row[];
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int ofs = COL_PER_BLK * bid;

    if (tid > n)return;


    if (tid == 0) {
        for (size_t i = 0; i < COL_PER_BLK; i++)
            base_row[i] = m2[(base_row_index * m2_width) + (ofs + i)];
    }
    __syncthreads();

    size_t max_cols=min((2*n)-ofs,3);
//    size_t max_cols = 3;

    if (tid == base_row_index) {
        normalize_self(&m2[tid * m2_width], base_row, scale[tid], max_cols, ofs);
    } else
        normalize_row(&m2[tid * m2_width], base_row, scale[tid], max_cols, ofs);
}

__global__ void gje_scale_calc(double *m2d, size_t n, size_t current_row, double *scale) {
    size_t m2_width = 2 * n;
    unsigned int tid = threadIdx.x;

    __shared__ double diag;
    double base = 0;

    if (tid == current_row)
        diag = m2d[current_row * m2_width + current_row];
    else
        base = m2d[tid * m2_width + current_row];
    __syncthreads();

    if (tid == current_row)
        scale[tid] = diag;
    else
        scale[tid] = base / diag;
}

__global__ void gje_set_identity(double *m2d, size_t n) {
    unsigned int tid = threadIdx.x;
    size_t m2d_width = 2 * n;
    m2d[(tid * m2d_width) + (n + tid)] = 1;
}

int main(int argc, char **argv) {

    size_t n = 0;
    int mode = FROM_FILE;
    string path;

    // arg parsing
    for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-') {
            switch (argv[i][1]) {
                case 'n':
                    n = strtoul(argv[++i], nullptr, 0);
                    break;
                case 'r':
                    mode = RANDOM;
                    break;
                case 'f':
                    mode = FROM_FILE;
                    path = argv[++i];
                    break;
                default:
                    cout << "invalid arguments";
            }
        }
    }
    double **m2_h = mxalloc(n, n, malloc);
    double **inv_h = mxalloc(n, n, malloc);
    if (mode == RANDOM) {
        fill_random(n, m2_h, pair<float, float>(-1e6, 1e6));
    } else {
        get_from_file(n, m2_h, path);
    }
    print_matrix(m2_h, n, n);

    dim3 block_dim(BLOCK_DIM);
    dim3 grid_dim((2 * n) / COL_PER_BLK + ((2 * n) % COL_PER_BLK != 0));

    size_t m2_width = 2 * n;
    double *m2_d = nullptr, *scale_d = nullptr;
    int error = 0;
    error |= hipMalloc((void **) &m2_d, n * m2_width * sizeof(double));
    for (size_t i = 0; i < n; i++) {
        error |= hipMemcpy(m2_d + i * m2_width, m2_h[i], n * sizeof(double), hipMemcpyHostToDevice);
    }
    error |= hipMalloc((void **) &scale_d, n * sizeof(double));
    if (error != hipSuccess) {
        cout << "couldn't allocate memory in device";
        cout << hipGetErrorString((hipError_t) error);
    }
    gje_set_identity<<<dim3(1), block_dim>>>(m2_d, n);
    hipDeviceSynchronize();

//    // check identity matrix
//    for (size_t i = 0; i < n; ++i) {
//        error |= hipMemcpy(inv_h[i], &m2_d[i * m2_width + n], sizeof(double) * n, hipMemcpyDeviceToHost);
//    }
//    print_matrix(inv_h, n, n);
    double **temp2_h = mxalloc(n, 2 * n, malloc);

    for (size_t i = 0; i < n; i++) {

        gje_scale_calc<<<1, block_dim>>>(m2_d, n, i, scale_d);
        hipDeviceSynchronize();
        double *temp = (double *) malloc(sizeof(double) * n);
        error |= hipMemcpy(temp, scale_d, sizeof(double) * n, hipMemcpyDeviceToHost);
        for (int j = 0; j < n; ++j)cerr << std::setprecision(2) << temp[j] << "\t";
        cerr << "\n";

        // check matrix before
        for (size_t j = 0; j < n; ++j) {
            error |= hipMemcpy(temp2_h[j], &m2_d[j * m2_width], sizeof(double) * 2 * n, hipMemcpyDeviceToHost);
        }
        cerr << "print M before:\n";
        print_matrix(temp2_h, n, 2 * n);


        gje_inverse<<<grid_dim, block_dim, COL_PER_BLK * sizeof(double)>>>(m2_d, n, i, scale_d);
        hipDeviceSynchronize();

        // check matrix
        for (size_t j = 0; j < n; ++j) {
            error |= hipMemcpy(temp2_h[j], &m2_d[j * m2_width], sizeof(double) * 2 * n, hipMemcpyDeviceToHost);
        }
        cerr << "print M:\n";
        print_matrix(temp2_h, n, 2 * n);
        cerr << "\n\n\n";


    }

    for (size_t i = 0; i < n; ++i) {
        error |= hipMemcpy(inv_h[i], &m2_d[i * m2_width + n], sizeof(double) * n, hipMemcpyDeviceToHost);
    }
    if (error != hipSuccess) {
        cout << "couldn't retrieve result";
        cout << hipGetErrorString((hipError_t) error);
    }
    print_matrix(inv_h, n, n);
    cout << inverse_test(m2_h, inv_h, n);
    mxfree(m2_h, n, free);
    mxfree(inv_h, n, free);
}
