#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include "matrix_utils.cpp"
#include <hip/hip_runtime.h>
#include <iomanip>

#define RANDOM 1
#define FROM_FILE 2
#define WITH_CPU 1
#define WITH_GPU 2

#define BLOCK_DIM (1<<10)
#define COL_PER_BLK 5

#define min(a, b) ((a < b) ? a : b)
#define ceil(a, b) (a / b + (a % b != 0))

using namespace std;

__device__ void normalize_row(double *target_row, const double *base_row, double scale, size_t n, size_t offset) {
    for (size_t i = 0; i < n; i++) {
        double temp = target_row[i + offset] - (base_row[i] * scale);
//        target_row[i + offset] = (IS_ZERO(temp) ? 0 : temp);
        target_row[i + offset] = temp;
    }
}

__device__ void
normalize_self(double *self, double const *self_but_in_share_memory, double scale, size_t n, size_t offset) {
    for (size_t i = 0; i < n; i++) {
        self[i + offset] = self_but_in_share_memory[i] / scale;
    }
}


__global__ void gje_inverse(double *m2, size_t n, size_t base_row_index, double *scale) {
    size_t m2_width = 2 * n;
    extern __shared__ double base_row[];
    unsigned int tid = threadIdx.x;
    unsigned int ofs = COL_PER_BLK * blockIdx.x;

    if (tid >= n)
        return;

    if (tid == 0)
        for (size_t i = 0; i < COL_PER_BLK; i++)
            base_row[i] = m2[(base_row_index * m2_width) + (ofs + i)];
    __syncthreads();

    size_t num_cols = min(m2_width - ofs, COL_PER_BLK);
    size_t step = blockDim.x;
    while (tid < n) {

        if (tid == base_row_index)
            normalize_self(&m2[tid * m2_width], base_row, scale[tid], num_cols, ofs);
        else
            normalize_row(&m2[tid * m2_width], base_row, scale[tid], num_cols, ofs);

        tid += step;
    }
}


__global__ void gje_scale_calc(const double *m2d, size_t n, size_t current_row, double *scale) {
    unsigned int tid = threadIdx.x;
    __shared__ double diag;
    size_t m2d_width = 2 * n;
    double base = 0;

    if (tid >= n)
        return;

    if (tid == 0)
        diag = m2d[current_row * m2d_width + current_row];
    __syncthreads();

    size_t step = blockDim.x;
    while (tid < n) {

        if (tid == current_row)
            scale[tid] = diag;
        else {
            base = m2d[tid * m2d_width + current_row];
            scale[tid] = base / diag;
        }
        tid += step;
    }


}

void cuda_check_err(const string &msg) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        cerr << msg << ":" << endl << hipGetErrorString((hipError_t) error) << endl;
        hipDeviceReset();
        exit(1);
    }
}

// ** num of threads per block = COL_PER_BLOCK
__global__ void gje_set_identity(double *m2d, size_t n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x + tid;
    size_t m2d_width = n * 2;

    if (idx >= n)
        return;

    m2d[idx * m2d_width + (n + idx)] = 1.0;
}

__host__ void gpu_inverse(double **matrix, size_t n, double **inverse, float *runtime) {
    size_t m2_width = 2 * n;
    double *m2_d = nullptr, *scale_d = nullptr;

    hipMalloc((void **) &m2_d, n * m2_width * sizeof(double));
    cuda_check_err("couldn't allocate memory in device");

    for (size_t i = 0; i < n; i++) {
        hipMemcpy(&m2_d[i * m2_width], matrix[i], n * sizeof(double), hipMemcpyHostToDevice);
        cuda_check_err("couldn't copy data from host to device");
    }

    hipMalloc((void **) &scale_d, n * sizeof(double));
    cuda_check_err("couldn't allocate memory in device");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    gje_set_identity<<<ceil(n, COL_PER_BLK), COL_PER_BLK>>>(m2_d, n);
    hipDeviceSynchronize();
    cuda_check_err("error in set_identity");

    for (size_t i = 0; i < n; i++) {
        stringstream str_i;
        gje_scale_calc<<<1, BLOCK_DIM>>>(m2_d, n, i, scale_d);
        hipDeviceSynchronize();
        str_i.str(string());
        str_i << "iter " << i << ") error in scale_calc";
        cuda_check_err(str_i.str());

        gje_inverse<<<ceil(m2_width, COL_PER_BLK), BLOCK_DIM, COL_PER_BLK * sizeof(double)>>>(m2_d, n, i, scale_d);
        hipDeviceSynchronize();
        str_i.str(string());
        str_i << "iter " << i << ") error in cpu_inverse";
        cuda_check_err(str_i.str());
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(runtime, start, stop);

    for (size_t i = 0; i < n; ++i) {
        hipMemcpy(inverse[i], &m2_d[i * m2_width + n], sizeof(double) * n, hipMemcpyDeviceToHost);
        cuda_check_err("couldn't copy data from host to host");
    }

    hipFree(m2_d);
    hipFree(scale_d);
}

int main(int argc, char **argv) {
    size_t n = 0;
    int get_mode = RANDOM;
    int exec_mode = WITH_GPU;
    string in_path;
    string out_path;

    // arg parsing
    for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-') {
            switch (argv[i][1]) {
                case 'n':
                    n = strtoul(argv[++i], nullptr, 0);
                    break;
                case 'r':
                    get_mode = RANDOM;
                    break;
                case 'f':
                    get_mode = FROM_FILE;
                    in_path = argv[++i];
                    break;
                case 'g':
                    exec_mode = WITH_GPU;
                    break;
                case 'c':
                    exec_mode = WITH_CPU;
                    break;
                case 'o':
                    out_path = argv[++i];
                    break;
                default:
                    cout << "invalid arguments";
            }
        }
    }
    double **matrix = mxalloc(n, n, malloc);
    double **inverse = mxalloc(n, n, malloc);
    float runtime = 0.0;

    if (get_mode == RANDOM)
        fill_random(n, matrix, pair<float, float>(-1e6, 1e6));
    else
        get_from_file(n, matrix, in_path);

    if (exec_mode == WITH_GPU)
        gpu_inverse(matrix, n, inverse, &runtime);
    else
        cpu_inverse(matrix, n, inverse, &runtime);

    if (!out_path.empty())
        save_to_file(n, inverse, out_path);

    cout << "time: " << runtime << "(ms)" << endl << "err: " << inverse_test(matrix, inverse, n) << endl;

    mxfree(matrix, n, free);
    mxfree(inverse, n, free);
    hipDeviceReset();
    return 0;
}
