#include "hip/hip_runtime.h"
#include <iostream>
#include<functional>
#include "matrix_utils.cpp"
#include <hip/hip_runtime.h>

#define RANDOM 1
#define FROM_FILE 2

#define BLOCK_DIM 1<<10
#define COL_P_BLK 5
using namespace std;

__device__ void normalize_row(const double *base_row, double *target_row, double scale, size_t n, size_t base_offset,
                              size_t target_offset) {
    for (size_t i = 0; i < n; i++) {
        target_row[i + target_offset] -= base_row[i + base_offset] * scale;
    }
}

__device__ void normalize_self(double *self, double scale, size_t n, size_t offset) {
    for (size_t i = 0; i < n; i++) {
        self[i + offset] /= scale;
    }
}

__global__ void gje_inverse(double **m, size_t n, size_t cr, double *scl) {
    extern __shared__ double mr[];
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int ofs = COL_P_BLK * bid;

    if (tid == 0) {
        for (size_t i = 0; i < COL_P_BLK; ++i)
            mr[i] = m[cr][ofs + i];
    }
    __syncthreads();
    if (tid == cr) {
        normalize_self(m[tid], scl[tid], COL_P_BLK, ofs);
    } else
        normalize_row(mr, m[tid], scl[tid], COL_P_BLK, 0, ofs);
}

__global__ void gje_scale_calc(double **m, size_t n, size_t cr, double *scl) {
    unsigned int tid = threadIdx.x;
    __shared__ double diag;

    if (tid == 0)
        diag = m[cr][cr];
    __syncthreads();

    if (tid == cr)
        scl[tid] = diag;
    else
        scl[tid] = m[tid][cr] / diag;
}

int main(int argc, char **argv) {
    size_t n = 0;
    int mode = FROM_FILE;
    string path;

    // arg parsing
    for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-') {
            switch (argv[i][1]) {
                case 'n':
                    n = strtoul(argv[++i], nullptr, 0);
                    break;
                case 'r':
                    mode = RANDOM;
                    break;
                case 'f':
                    mode = FROM_FILE;
                    path = argv[++i];
                    break;
                default:
                    cout << "invalid arguments";
            }
        }
    }
    double **m_h = mxalloc(n, n, malloc);
    double **inv_h = mxalloc(n, n, malloc);
    if (mode == RANDOM) {
        fill_random(n, m_h, pair<float, float>(-1e6, 1e6));
    } else {
        get_from_file(n, m_h, path);
    }
    print_matrix(m_h, n, n);

    double **m_d = nullptr, *scl_d = nullptr;
    hipMalloc((void **) &scl_d, n * sizeof(double));
    hipMalloc((void **) &m_d, n * sizeof(double *));
    for (size_t i = 0; i < n; ++i) {
        hipError_t err = hipMalloc((void **) &m_d[i], 2 * n * sizeof(double));
        cout << hipGetErrorString(err);
        hipMemcpy(&m_d[i], m_h[i], n * sizeof(double), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        m_d[i][n + i] = 1;
    }

    unsigned int grid_dim = (2 * n) / COL_P_BLK + ((2 * n) % COL_P_BLK != 0);
    dim3 BL(BLOCK_DIM);
    dim3 GR(grid_dim);
    for (size_t i = 0; i < n; ++i) {
        gje_scale_calc<<<1, BL>>>(m_d, n, i, scl_d);
        gje_inverse<<<GR, BL, COL_P_BLK * sizeof(double)>>>(m_d, n, i, scl_d);
        hipDeviceSynchronize();
    }
    for (size_t i = 0; i < n; ++i) {
        hipMemcpy(&inv_h[i], &m_d[i][n], sizeof(double) * n, hipMemcpyDeviceToHost);
    }
    mxfree(m_h, n, free);
    mxfree(inv_h, n, free);
    print_matrix(inv_h, n, n);
    cout << inverse_test(m_h, inv_h, n);
}
